
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <map>
#include <string>
#include <fstream>
#include <vector>

using namespace std;

#define ll long long

const int BLOCK_SIZE = 2;
const int GRID_SIZE = 32;

// Use naive method
__device__ bool isPrime(ll n)
{
    if(n<2)
        return false;
        
    for(ll i=2;i*i<=n;i++)
        if(n%i==0)
            return false; 

    return true;
}

// Read numbers from file and add to vector
std::vector<ll> reaadFile(char* arg){
    vector<ll> numbersFromFile;
    std::ifstream infile(arg);
    ll number;

    while (infile >> number) {
        numbersFromFile.push_back(number);
    }

    return numbersFromFile;
}

__global__ void calculate(ll *Arr, bool *results, int sizeOfArray){

    long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < sizeOfArray && y < sizeOfArray) 
	{
		for (long k = 0; k < sizeOfArray; k++) {
			results[x + sizeOfArray * y] += isPrime(Arr[x + sizeOfArray * y]);
		}
	}

}
    
int main(int argc, char** argv )
{
    float time;

    if ( argc != 2 )
    {
        printf("Pass file path\n");
        return -1;
    }

    vector<ll> numbersFromFile = reaadFile(argv[1]);

    int sizeOfArray = numbersFromFile.size();
	
    ll numbersFromFileArr[sizeOfArray];
    std::copy(numbersFromFile.begin(), numbersFromFile.end(), numbersFromFileArr);

    unsigned int i;
    bool results[sizeOfArray];

    ll* c_arr;
    bool* c_results;

    hipMemcpy(c_arr, numbersFromFileArr, sizeOfArray, hipMemcpyHostToDevice);
    hipMalloc((void**) &c_results, sizeOfArray);

    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grids(GRID_SIZE, GRID_SIZE);

    //Start timer
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    //results and numberFromFileArr to send into kernel


    calculate<<<blocks, grids>>>(c_arr, c_results, sizeOfArray);

    //End timer and put result into time variable
    hipDeviceSynchronize();			 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

    
    printf("Czas: %.4fms\n", time);

    if (hipMemcpy(results, c_results , sizeOfArray, hipMemcpyDeviceToHost) != hipSuccess) {
		cout<<"GPU to CPU copy error\n";
	}

    hipFree(c_arr);
    hipFree(c_results);

    for (int i = 0; i < sizeOfArray; i++){
        if (results[i]){
            cout << numbersFromFileArr[i] << " prime" << endl;
        } else {
            cout << numbersFromFileArr[i] << " composite" << endl;
        }
    }

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <map>
#include <string>
#include <fstream>
#include <vector>

using namespace std;

#define ll long long

const int BLOCK_SIZE = 2;
const int GRID_SIZE = 1;

// Use naive method
__device__ bool isPrime(ll n)
{
    printf("IS PRIME FUNCTION %lld \n", n);
    if(n<2)
        return false;
        
    for(ll i=2;i*i<=n;i++)
        if(n%i==0)
            return false; 

    printf("%lld IS PRIME \n", n);
    return true;
}

// Read numbers from file and add to vector
std::vector<ll> reaadFile(char* arg){
    vector<ll> numbersFromFile;
    std::ifstream infile(arg);
    ll number;

    while (infile >> number) {
        numbersFromFile.push_back(number);
    }

    return numbersFromFile;
}

__global__ void calculate(ll *Arr, bool *results, int sizeOfArray){

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;

    printf("X: %d \n", x);
    printf("BLOCK_ID: %d\n", blockIdx.x);
    printf("BLOCK_DIM: %d \n", blockDim.x);
    printf("ThreaD_ID: %d\n", threadIdx.x);

	if (x < sizeOfArray) 
	{
        results[x] += isPrime(Arr[x]);
	}

}

bool isPrimeMain(ll n){
    if(n<2)
        return false;
        
    for(ll i=2;i*i<=n;i++)
        if(n%i==0)
            return false; 

    return true;
}
    
int main(int argc, char** argv )
{
    float time;

    if ( argc != 2 )
    {
        printf("Pass file path\n");
        return -1;
    }

    vector<ll> numbersFromFile = reaadFile(argv[1]);

    int sizeOfArray = numbersFromFile.size();
    int sizeToAllocateLongLong = sizeOfArray * sizeof(ll);
    int sizeToAllocateBool = sizeOfArray * sizeof(bool);

	
    ll numbersFromFileArr[sizeOfArray];
    std::copy(numbersFromFile.begin(), numbersFromFile.end(), numbersFromFileArr);

    cout << "BEFORE ALLOCATE RESULTS" << endl;

    unsigned int i;
    bool* results = (bool *) malloc (sizeToAllocateBool);

    ll* c_arr;
    bool* c_results;

    cout << "CUDA MALLOC" << endl;

    hipMalloc((void**) &c_arr, sizeToAllocateLongLong);
    hipMalloc((void**) &c_results, sizeToAllocateBool);

    cout << "CUDA MEMCPY" << endl;

    hipMemcpy((void *)c_arr, (void *)numbersFromFileArr, sizeToAllocateLongLong, hipMemcpyHostToDevice);

    cout << "BEFORE DIM3" << endl;

    dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grids(GRID_SIZE, GRID_SIZE);

    //Start timer
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    cout << "BEFORE CALL KERNEL: " << endl;
    calculate<<<sizeOfArray, GRID_SIZE>>>(c_arr, c_results, sizeOfArray);

    //End timer and put result into time variable
    hipDeviceSynchronize();			 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

    printf("Czas: %.4fms\n", time);

    if (hipMemcpy((void *)results, (void *)c_results , sizeToAllocateBool, hipMemcpyDeviceToHost) != hipSuccess) {
		cout<<"GPU to CPU copy error\n";
	}

    hipFree(c_arr);
    hipFree(c_results);



    for (int i = 0; i < sizeOfArray; i++){
        if (results[i]){
            cout << numbersFromFileArr[i] << " prime";
        } else {
            cout << numbersFromFileArr[i] << " composite";
        }

        if (isPrimeMain(numbersFromFileArr[i]) == results[i])
            cout << " - GOOD" << endl;
    }

    free(results);
    return 0;
}
